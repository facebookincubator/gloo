/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "gloo/nccl/nccl.h"

#include <algorithm>
#include <unordered_map>

#include "gloo/cuda_private.h"

namespace gloo {
namespace nccl {

// Allocate a set of per-device streams used to serialize NCCL op scheduling.
// These ensure concurrent NCCL ops are not interleaved across devices (i.e.,
// through priority scheduling), resulting in deadlock. Use a function-scope
// static to avoid SIOF with the CUDA runtime.
static CudaDeviceStreams& getNcclStreams() {
  static CudaDeviceStreams ncclStreams;
  return ncclStreams;
}

template <typename T>
class NCCLContext {
 public:
  NCCLContext(const std::vector<int>& devices) : devices(devices) {
    // Initialze comms. Synchronize with conflicting CUDA and NCCL operations.
    comms.resize(devices.size());
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    NCCL_CHECK(ncclCommInitAll(comms.data(), devices.size(), devices.data()));
  }
  ~NCCLContext() {
    /*
     * TODO(T30279827) Temporarily disable calling ncclCommDestroy
     * Calling ncclCommDestroy while program exiting is undefined
     * according to nvidia, and lead to segfault in NCCL 2
     * (whether it is called before or after the CUDA runtime destructor).
     * Temporarily disable it in destructor to avoid segfault.
     * Following up with Nvidia for long term solution.
     */

    /*
    for (auto i = 0; i < devices.size(); ++i) {
      CudaDeviceScope scope(devices[i]);
      {
        // Synchronize memory allocation with NCCL operations
        std::lock_guard<std::mutex> lock(CudaShared::getMutex());
        ncclCommDestroy(comms[i]);
      }
    }
    */
  }

  // Instances cannot be copied or copy-assigned
  NCCLContext(const NCCLContext&) = delete;
  NCCLContext& operator=(const NCCLContext&) = delete;

  const std::vector<int> devices;
  std::vector<ncclComm_t> comms;
};

// Initializing NCCL communications is expensive. Allocate context as needed per
// unique device set and cache for reuse.
template <typename T>
static std::shared_ptr<NCCLContext<T>> getNcclContext(
    const NCCLExecution<T>& ex) {
  static std::unordered_map<std::string, std::shared_ptr<NCCLContext<T>>>
      contexts;
  const auto key = ex.getKey();
  {
    static std::mutex m;
    std::lock_guard<std::mutex> lock(m);
    if (!contexts[key]) {
      contexts[key] = std::make_shared<NCCLContext<T>>(ex.getDevices());
    }
  }
  const auto context = contexts[key];
  GLOO_ENFORCE_NE(context.get(), (void*)nullptr);
  return context;
}

template <typename T>
NCCLExecution<T>::NCCLExecution(std::vector<NCCLElement<T>>&& elements)
    : elements(std::move(elements)) {
  // Allocate events to synchronize source, destination, and NCCL streams
  ncclEvents.resize(this->elements.size());
  for (auto i = 0; i < this->elements.size(); i++) {
    CudaDeviceScope scope(this->elements[i].device);
    CUDA_CHECK(hipEventCreateWithFlags(
        &ncclEvents[i], hipEventDefault | hipEventDisableTiming));
  }
}

template <typename T>
NCCLExecution<T>::~NCCLExecution() noexcept(false) {
  for (auto i = 0; i < this->elements.size(); i++) {
    CudaDeviceScope scope(this->elements[i].device);
    CUDA_CHECK(hipEventDestroy(ncclEvents[i]));
  }
}

template <typename T>
std::vector<int> NCCLExecution<T>::getDevices() const {
  std::vector<int> result;
  result.reserve(elements.size());
  for (const auto& el : elements) {
    GLOO_ENFORCE(
        // Performing a linear search given small set of devices
        std::find(result.begin(), result.end(), el.device) == result.end(),
        "NCCL elements must map to unique devices");
    result.push_back(el.device);
  }
  return result;
}

template <typename T>
std::string NCCLExecution<T>::getKey() const {
  // Construct a key representing the order-dependent devices in this NCCL
  // execution. This is used to index into the NCCL context map and allows an
  // implicit association between elements[i].device and NCCLContext::comms[i]
  std::string result;
  for (const auto& el : elements) {
    result += std::to_string(el.device) + ",";
  }
  return result;
}

template <typename T>
class ncclTypeWrapper;

template <>
class ncclTypeWrapper<int8_t> {
 public:
  static const ncclDataType_t type = ncclChar;
};

template <>
class ncclTypeWrapper<uint8_t> {
 public:
  static const ncclDataType_t type = ncclChar;
};

template <>
class ncclTypeWrapper<int32_t> {
 public:
  static const ncclDataType_t type = ncclInt;
};

template <>
class ncclTypeWrapper<int64_t> {
 public:
  static const ncclDataType_t type = ncclInt64;
};

template <>
class ncclTypeWrapper<uint64_t> {
 public:
  static const ncclDataType_t type = ncclUint64;
};

template <>
class ncclTypeWrapper<float16> {
 public:
  static const ncclDataType_t type = ncclHalf;
};

template <>
class ncclTypeWrapper<float> {
 public:
  static const ncclDataType_t type = ncclFloat;
};

template <>
class ncclTypeWrapper<double> {
 public:
  static const ncclDataType_t type = ncclDouble;
};

template <typename T>
NCCLOp<T>::NCCLOp(NCCLExecution<T>&& execution)
    : execution_(std::move(execution)), context_(getNcclContext(execution_)) {}

template <typename T>
void NCCLOp<T>::wait() {
  auto& elements = execution_.elements;
  for (auto i = 0; i < elements.size(); ++i) {
    CudaDeviceScope scope(elements[i].device);
    elements[i].dstStream.wait();
  }
}

template <typename T>
template <typename F>
void NCCLOp<T>::runNCCL(F&& f) {
  const auto& elements = execution_.elements;
  const auto& ncclEvents = execution_.ncclEvents;
  const auto& comms = context_->comms;

  // Synchronize memory allocation with NCCL operations
  std::lock_guard<std::mutex> lock(CudaShared::getMutex());

#if NCCL_VERSION_MIN(2,0,0)
  NCCL_CHECK(ncclGroupStart());
#endif
  // Kick off the NCCL operation on each device
  for (auto i = 0; i < elements.size(); i++) {
    const auto& element = elements[i];
    const auto& srcStream = element.srcStream.getStream();
    const auto& dstStream = element.dstStream.getStream();
    const auto& ncclStream = getNcclStreams()[element.device];
    const auto& srcEvent = element.srcStream.getEvent();
    const auto& dstEvent = element.dstStream.getEvent();

    CudaDeviceScope scope(element.device);
    // Synchronize the source and destination with the NCCL stream. Record
    // events in the source and destination streams, and wait on these in the
    // NCCL streams.
    CUDA_CHECK(hipEventRecord(srcEvent, srcStream));
    CUDA_CHECK(hipStreamWaitEvent(ncclStream, srcEvent, 0));
    if (srcStream != dstStream) {
      CUDA_CHECK(hipEventRecord(dstEvent, dstStream));
      CUDA_CHECK(hipStreamWaitEvent(ncclStream, dstEvent, 0));
    }
    // Run the operation
    f(element, comms[i], ncclStream);
  }
#if NCCL_VERSION_MIN(2,0,0)
  NCCL_CHECK(ncclGroupEnd());
#endif
  for (auto i = 0; i < elements.size(); ++i) {
    const auto& element = elements[i];
    const auto& ncclStream = getNcclStreams()[element.device];
    const auto& dstStream = element.dstStream.getStream();
    const auto& dstEvent = element.dstStream.getEvent();

    CudaDeviceScope scope(element.device);
    // Record an event in the NCCL stream signaling the operation is complete.
    // Synchronize with the destination stream.
    CUDA_CHECK(hipEventRecord(ncclEvents[i], ncclStream));
    CUDA_CHECK(hipStreamWaitEvent(dstStream, ncclEvents[i], 0));
    CUDA_CHECK(hipEventRecord(dstEvent, dstStream));
  }
}

template <typename T>
void ReduceOp<T>::runAsync() {
  const auto op = op_;
  const auto root = root_;
  this->runNCCL([op, root](
      const NCCLElement<T>& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclReduce(
        *element.src,
        *element.dst,
        element.src.getCount(),
        ncclTypeWrapper<T>::type,
        op,
        root,
        comm,
        stream));
  });
}

template <typename T>
void AllreduceOp<T>::runAsync() {
  const auto op = op_;
  this->runNCCL([op](
      const NCCLElement<T>& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclAllReduce(
        *element.src,
        *element.dst,
        element.src.getCount(),
        ncclTypeWrapper<T>::type,
        op,
        comm,
        stream));
  });
}

template <typename T>
void ReduceScatterOp<T>::runAsync() {
  const auto op = op_;
  this->runNCCL([op](
      const NCCLElement<T>& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclReduceScatter(
        *element.src,
        *element.dst,
        element.dst.getCount(),
        ncclTypeWrapper<T>::type,
        op,
        comm,
        stream));
  });
}

template <typename T>
void BroadcastOp<T>::runAsync() {
  const int root = root_;
  this->runNCCL([root](
      const NCCLElement<T>& element, ncclComm_t comm, hipStream_t stream) {
    NCCL_CHECK(ncclBcast(
        *element.dst,
        element.dst.getCount(),
        ncclTypeWrapper<T>::type,
        root,
        comm,
        stream));
  });
}

template <typename T>
void AllgatherOp<T>::runAsync() {
  this->runNCCL([](
      const NCCLElement<T>& element, ncclComm_t comm, hipStream_t stream) {
#if NCCL_VERSION_MIN(2,0,0)
    NCCL_CHECK(ncclAllGather(
        *element.src,
        *element.dst,
        element.src.getCount(),
        ncclTypeWrapper<T>::type,
        comm,
        stream));
#else
    NCCL_CHECK(ncclAllGather(
        *element.src,
        element.src.getCount(),
        ncclTypeWrapper<T>::type,
        *element.dst,
        comm,
        stream));
#endif
  });
}

#define DEFINE_NCCL_TYPES_AND_OPS(T)                                    \
template class NCCLExecution<T>;                                        \
template class NCCLContext<T>;                                          \
template class NCCLOp<T>;                                               \
                                                                        \
template class ReduceOp<T>;                                             \
template class AllreduceOp<T>;                                          \
template class ReduceScatterOp<T>;                                      \
template class BroadcastOp<T>;                                          \
template class AllgatherOp<T>;

DEFINE_NCCL_TYPES_AND_OPS(int8_t);
DEFINE_NCCL_TYPES_AND_OPS(uint8_t);
DEFINE_NCCL_TYPES_AND_OPS(int32_t);
DEFINE_NCCL_TYPES_AND_OPS(int64_t);
DEFINE_NCCL_TYPES_AND_OPS(uint64_t);
DEFINE_NCCL_TYPES_AND_OPS(float16);
DEFINE_NCCL_TYPES_AND_OPS(float);
DEFINE_NCCL_TYPES_AND_OPS(double);

} // namespace nccl
} // namespace gloo
