#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include "gloo/cuda_private.h"

#include <array>

#include <hip/hip_runtime.h>

// Disable strict aliasing errors for CUDA 9.
#if CUDA_VERSION >= 9000
#ifdef __GNUC__
#if __GNUC__ > 4 || (__GNUC__ == 4 && __GNUC_MINOR__ >= 6)
#pragma GCC diagnostic push
#endif
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#endif // __GNUC__
#endif // CUDA_VERSION >= 9000
#include <hip/hip_fp16.h>
#if CUDA_VERSION >= 9000
#ifdef __GNUC__
#if __GNUC__ > 4 || (__GNUC__ == 4 && __GNUC_MINOR__ >= 6)
#pragma GCC diagnostic pop
#endif
#endif // __GNUC__
#endif // CUDA_VERSION >= 9000

#include "gloo/common/common.h"
#include "gloo/types.h"

namespace gloo {

template<typename T>
__global__ void initializeMemory(
    T* ptr,
    const int val,
    const size_t count,
    const size_t stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for (; i < count; i += blockDim.x) {
    ptr[i] = (i * stride) + val;
  }
}

template<>
__global__ void initializeMemory<float16>(
    float16* ptr,
    const int val,
    const size_t count,
    const size_t stride) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  half* ptrAsHalf = (half*) ptr;
  for (; i < count; i += blockDim.x) {
    ptrAsHalf[i] = __float2half(static_cast<float>((i * stride) + val));
  }
}

template<typename T>
CudaMemory<T>::CudaMemory(size_t elements)
    : elements(elements),
      bytes(elements * sizeof(T)) {
  CUDA_CHECK(hipGetDevice(&device_));
  // Sychronize memory allocation with NCCL operations
  std::lock_guard<std::mutex> lock(CudaShared::getMutex());
  CUDA_CHECK(hipMalloc(&ptr_, bytes));
}

template<typename T>
CudaMemory<T>::CudaMemory(CudaMemory<T>&& other) noexcept
  : elements(other.elements),
    bytes(other.bytes),
    device_(other.device_),
    ptr_(other.ptr_) {
  // Nullify pointer on move source
  other.ptr_ = nullptr;
}

template<typename T>
CudaMemory<T>::~CudaMemory() noexcept(false) {
  CudaDeviceScope scope(device_);
  if (ptr_ != nullptr) {
    // Sychronize memory allocation with NCCL operations
    std::lock_guard<std::mutex> lock(CudaShared::getMutex());
    CUDA_CHECK(hipFree(ptr_));
  }
}

// Instantiate template
template class CudaMemory<float>;
template class CudaMemory<float16>;

// Lookup PCI bus IDs for device.
// As the number of available devices won't change at
// runtime we can seed this cache on the first call.
const std::string& getCudaPCIBusID(int device) {
  static std::once_flag once;
  static std::vector<std::string> busIDs;

  std::call_once(once, [](){
    std::array<char, 16> buf;
    auto count = getDeviceCount();
    busIDs.resize(count);
    for (auto i = 0; i < count; i++) {
      CUDA_CHECK(hipDeviceGetPCIBusId(buf.data(), buf.size(), i));
      busIDs[i] = buf.data();
    }
  });

  return busIDs[device];
}

} // namespace gloo
